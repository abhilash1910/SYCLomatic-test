#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>

void matmul_forward_cublaslt(float *out, const float *input, const float *weight, const float *bias, int B, int T, int C, int OC){

  int has_bias = (bias!=NULL);
  int has_gelu =0;
  
  
  if((uintptr_t)bias % 16 ==0){
    printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
    exit(EXIT_FAILURE);
  }
  
  int returnedResults = 0;
  hipblasLtMatmulDesc_t operationDesc;
  hipblasLtMatmulPreference_t preference;
  hipblasLtMatrixLayout_t inputLayout;
  hipblasLtMatrixLayout_t weightLayout;
  hipblasLtMatrixLayout_t biasLayout;
  hipblasLtMatrixLayout_t outputLayout;
  hipblasLtMatmulHeuristicResult_t heuristic;
  
  hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
  hipblasOperation_t opTranspose = HIPBLAS_OP_T;
  hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_DEFAULT;
  if (has_bias && has_gelu) {
      epilogueBias = HIPBLASLT_EPILOGUE_GELU_BIAS;
  } else if (has_bias) {
      epilogueBias = HIPBLASLT_EPILOGUE_BIAS;
  } else if (has_gelu) {
      epilogueBias = HIPBLASLT_EPILOGUE_GELU;
  }
  cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute_type, HIP_R_32F));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

  // define matrix layouts
  cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, HIP_R_32F, C, OC, C));
  cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, HIP_R_32F, C, B*T, C));
  cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, HIP_R_32F, OC, B*T, OC));
  cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, HIP_R_32F, OC, 1, OC));

  // create a preference handle with specified max workspace
  cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
  cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
      HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
      &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

  // find a suitable algorithm
  cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
      weightLayout, inputLayout, outputLayout, outputLayout,
      preference, 1, &heuristic, &returnedResults));
  if (returnedResults == 0) {
      printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d, gelu: %d\n",
          B, T, C, OC, has_bias, has_gelu);
      exit(EXIT_FAILURE);
  }

  // call the matmul
  const float alpha = 1.0f, beta = 0.0f;
  cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
      &alpha, weight, weightLayout, inp, inputLayout, &beta,
      out, outputLayout, out, outputLayout, &heuristic.algo,
      cublaslt_workspace, cublaslt_workspace_size, 0));

  // cleanups
  cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
  cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
  cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}


void matmul_forward(float* out,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C, int OC){
                    
                    
  matmul_forward_cublaslt(out, inp, weight, bias, B, T, C, OC);                    

}

int main(int argc, char **argv) {
    srand(0);

    int B = 32;
    int T = 1024;
    int C = 768;
    int OC = 768 * 4; // expansion of 4, e.g. in the MLP

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * OC * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(OC * C);
    float* bias = make_random_float(OC);

    // move to GPU
    float* d_out;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, OC * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * OC * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, OC * sizeof(float), hipMemcpyHostToDevice));
    matmul_forward(out, inp, weight, bias, B, T, C, OC);   
    
    // free memory
    free(out);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    
    return 0;
}

