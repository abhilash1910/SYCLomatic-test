#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipblaslt.h>

//global vars for cublaslt
const size_t cublaslt_workspace_size = 32 * 1024 * 1024;
void* cublaslt_workspace = NULL;
hipblasComputeType_t cublas_compute_type = HIPBLAS_COMPUTE_32F;
hipblasLtHandle_t cublaslt_handle;
hipblasHandle_t cublas_handle;



void cuda_check(hipError_t error, const char *file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};
#define cudaCheck(err) (cuda_check(err, __FILE__, __LINE__))


void cublasCheck(hipblasStatus_t status, const char *file, int line)
{
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("[cuBLAS ERROR]: %d %s %d\n", status, file, line);
        exit(EXIT_FAILURE);
    }
}
#define cublasCheck(status) { cublasCheck((status), __FILE__, __LINE__); }



float* make_fixed_float(size_t n){
  float* arr = (float*)malloc(n * sizeof(float));
  for(int i=0;i<n;i++){
    arr[i] = 1.0f;
  }
  return arr;
}

void matmul_forward_cublaslt(float *out, const float *inp, const float *weight, const float *bias, int B, int T, int C, int OC){

  int has_bias = (bias!=NULL);
  int has_gelu =0;
  
  
  if((uintptr_t)bias % 16 !=0){
    printf("Bias pointer is not aligned (cuBLASLt requirement)!\n");
    exit(EXIT_FAILURE);
  }
  
  int returnedResults = 0;
  hipblasLtMatmulDesc_t operationDesc;
  hipblasLtMatmulPreference_t preference;
  hipblasLtMatrixLayout_t inputLayout;
  hipblasLtMatrixLayout_t weightLayout;
  hipblasLtMatrixLayout_t biasLayout;
  hipblasLtMatrixLayout_t outputLayout;
  hipblasLtMatmulHeuristicResult_t heuristic;
  
  hipblasOperation_t opNoTranspose = HIPBLAS_OP_N;
  hipblasOperation_t opTranspose = HIPBLAS_OP_T;
  hipblasLtEpilogue_t epilogueBias = HIPBLASLT_EPILOGUE_DEFAULT;
  if (has_bias && has_gelu) {
      epilogueBias = HIPBLASLT_EPILOGUE_GELU_BIAS;
  } else if (has_bias) {
      epilogueBias = HIPBLASLT_EPILOGUE_BIAS;
  } else if (has_gelu) {
      epilogueBias = HIPBLASLT_EPILOGUE_GELU;
  }
  cublasCheck(hipblasLtMatmulDescCreate(&operationDesc, cublas_compute_type, HIP_R_32F));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opTranspose, sizeof(opTranspose)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opNoTranspose, sizeof(opNoTranspose)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogueBias, sizeof(epilogueBias)));
  cublasCheck(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias)));

  // define matrix layouts
  cublasCheck(hipblasLtMatrixLayoutCreate(&weightLayout, HIP_R_32F, C, OC, C));
  cublasCheck(hipblasLtMatrixLayoutCreate(&inputLayout, HIP_R_32F, C, B*T, C));
  cublasCheck(hipblasLtMatrixLayoutCreate(&outputLayout, HIP_R_32F, OC, B*T, OC));
  cublasCheck(hipblasLtMatrixLayoutCreate(&biasLayout, HIP_R_32F, OC, 1, OC));

  // create a preference handle with specified max workspace
  cublasCheck(hipblasLtMatmulPreferenceCreate(&preference));
  cublasCheck(hipblasLtMatmulPreferenceSetAttribute(preference,
      HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
      &cublaslt_workspace_size, sizeof(cublaslt_workspace_size)));

  // find a suitable algorithm
  cublasCheck(hipblasLtMatmulAlgoGetHeuristic(cublaslt_handle, operationDesc,
      weightLayout, inputLayout, outputLayout, outputLayout,
      preference, 1, &heuristic, &returnedResults));
  if (returnedResults == 0) {
      printf("No cuBLASLt algorithm: B: %d, T: %d, C: %d, OC: %d, bias: %d, gelu: %d\n",
          B, T, C, OC, has_bias, has_gelu);
      exit(EXIT_FAILURE);
  }

  // call the matmul
  const float alpha = 1.0f, beta = 0.0f;
  cublasCheck(hipblasLtMatmul(cublaslt_handle, operationDesc,
      &alpha, weight, weightLayout, inp, inputLayout, &beta,
      out, outputLayout, out, outputLayout, &heuristic.algo,
      cublaslt_workspace, cublaslt_workspace_size, 0));

  // cleanups
  cublasCheck(hipblasLtMatmulPreferenceDestroy(preference));
  cublasCheck(hipblasLtMatmulDescDestroy(operationDesc));
  cublasCheck(hipblasLtMatrixLayoutDestroy(weightLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(inputLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(outputLayout));
  cublasCheck(hipblasLtMatrixLayoutDestroy(biasLayout));
}


void matmul_forward(float* out,
                    const float* inp, const float* weight, const float* bias,
                    int B, int T, int C, int OC){
                    
                    
  matmul_forward_cublaslt(out, inp, weight, bias, B, T, C, OC);                    

}

void validate_results(const float* kernel_result, int num_elements) {
    int nfaults = 0;
    for (int i = 0; i < num_elements; i++) {
        // print the first few comparisons
        if (kernel_result[i] == 769.0f) {
            printf("%f %f\n", kernel_result[i]);
        }
        else{
            nfaults++;
            if (nfaults >= 10) {
                exit(EXIT_FAILURE);
            }
        }
    }
    if (nfaults > 0) {
        exit(EXIT_FAILURE);
    }
    printf("OK\n");
}


int main(int argc, char **argv) {
    srand(0);

    int B = 32;
    int T = 1024;
    int C = 768;
    int OC = 768 * 4; // expansion of 4, e.g. in the MLP

    // set up the device
    int deviceIdx = 0;
    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipSetDevice(deviceIdx));
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceIdx);
    printf("Device %d: %s\n", deviceIdx, deviceProp.name);

    // setup cuBLAS and cuBLASLt
    cublasCheck(hipblasCreate(&cublas_handle));
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    // TF32 precision is equivalent to torch.set_float32_matmul_precision('high')
    int enable_tf32 = deviceProp.major >= 8 ? 1 : 0;
    printf("enable_tf32: %d\n", enable_tf32);
    cublas_compute_type = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    hipblasMath_t cublas_math_mode = enable_tf32 ? HIPBLAS_TF32_TENSOR_OP_MATH : HIPBLAS_DEFAULT_MATH;
    cublasCheck(hipblasSetMathMode(cublas_handle, cublas_math_mode));
    // setup the (global) cuBLASLt workspace
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));

    // create host memory of random numbers
    float* out = (float*)malloc(B * T * OC * sizeof(float));
    float* inp = make_fixed_float(B * T * C);
    float* weight = make_fixed_float(OC * C);
    float* bias = make_fixed_float(OC);

    // move to GPU
    float* d_out;
    float* d_inp;
    float* d_weight;
    float* d_bias;
    cudaCheck(hipMalloc(&d_out, B * T * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * OC * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, OC * sizeof(float)));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * OC * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, bias, OC * sizeof(float), hipMemcpyHostToDevice));
    matmul_forward(out, inp, weight, bias, B, T, C, OC);   
    validate_results(out, B * T * OC);
    
    // free memory
    free(out);
    free(inp);
    free(weight);
    free(bias);
    cudaCheck(hipFree(d_out));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasDestroy(cublas_handle));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    
    return 0;
}
